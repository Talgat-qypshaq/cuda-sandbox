#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void testFunction(float *dev_a, float *dev_b, float dev_c)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    printf("dev[%d] = %.2f;\n", thread, dev_a[thread]);
    printf("b = %.2f;\n", *dev_b);
    printf("c 1 = %.2f;\n", dev_c);
    dev_c = dev_c*dev_c;
    printf("c 2 = %.2f;\n", dev_c);
  }
}

int main()
{
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE START\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\nBEFORE END\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);
  float b = 25;
  float *dev_b;
  hipMalloc((void**)&dev_b, sizeof(float));
  hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);
  float c = 77;
  testFunction<<<BLOCKS, THREADS>>>(dev_a, dev_b, c);
  hipFree(dev_a);
  hipFree(dev_b);
  printf("after kernel free: c = %.2f;\n", c);
  return 0;
}
