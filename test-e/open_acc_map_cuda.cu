#include "hip/hip_runtime.h"
#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits>
#include <mpi.h>

#define NPP_MAXABS_64F ( 1.7976931348623158e+308 )
#define NPP_MINABS_64F ( 2.2250738585072014e-308 )
#define NPP_MAXABS_32F ( 3.402823466e+38f )
#define NPP_MINABS_32F ( 1.175494351e-38f )
#define VECL 8
#define WID 4
#define i_pcolumnv_cuda(j, k, k_block, num_k_blocks) ( ((j) / ( VECL / WID)) * WID * ( num_k_blocks + 2) + (k) + ( k_block + 1 ) * WID )
void fillArray(float *b);
using namespace std;
__device__ int slope_limiter_sb_1(int &l, int &m, int &r)
{
  int a = r-m;
  int b = m+l;
  printf("a = %d; b = %d; max = %d;\n", a, b, max(a, b));
  return max(a,b);
}
__device__ int slope_limiter_1(int &l,int &m,int &r)
{
   return slope_limiter_sb_1(l,m,r);
}
__device__ void compute_plm_coeff_1(int *values, int k, int a[2], float threshold)
{
  // scale values closer to 1 for more accurate slope limiter calculation
  float scale = 1./threshold;
  int v_1 = values[k - 1] * threshold;
  int v_2 = values[k] * threshold;
  int v_3 = values[k + 1] * threshold;
  //int d_cv = (v_1+v_2+v_3)*threshold;
  int d_cv = slope_limiter_1(v_1, v_2, v_3) * threshold;
  //int d_cv = slope_limiter_1(values[k-1]*scale, values[k]*scale, values[k+1]*scale)*threshold;
  printf("values[%d] = %d; d_cv = %d\n", k, values[k], d_cv);
  a[0] = values[k] - d_cv * 0.5;
  a[1] = d_cv * 0.5;
}
/*____________________________________________________________________________*/
__device__ int slope_limiter_sb_2(int &l, int &m, int &r)
{
  int a = r-m;
  int b = m+l;
  printf("a = %d; b = %d; max = %d;\n", a, b, max(a, b));
  return max(a,b);
}
__device__ int slope_limiter_2(int &l)
{
    return 2;
    //return slope_limiter_sb_2(l,m,r);
}
__device__ void compute_plm_coeff_2(int *values, int k, int a[2], float threshold)
{
  //scale values closer to 1 for more accurate slope limiter calculation
  float scale = 1./threshold;
  int v_1 = values[k - 1] * scale;
  int v_2 = values[k] * scale;
  int v_3 = values[k + 1] * scale;
  //int d_cv = (v_1+v_2+v_3)*threshold;
  //int d_cv = slope_limiter(v_1, v_2, v_3) * threshold;
  int d_cv = slope_limiter_2(v_1);
  printf("values[%d] = %d; d_cv = %d\n", k, values[k], d_cv);
  a[0] = values[k] - d_cv * 0.5;
  a[1] = d_cv * 0.5;
}
/*____________________________________________________________________________*/
__global__ void cudaFunction(float *b)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if(index == 0)
  {
    //float *a = &b[index];
    //printf( "size of a = %ld;\n", sizeof(*a) );
    //a[index+1] = CUDASIZE*CUDASIZE - index;
    int a[2];
    int j = 2;
    int k = 1;
    float minValue = 4.3;
    float nblocks = 5.1;
    float r1 = i_pcolumnv_cuda(j, 0, -1, nblocks);
    int *v = new int[2];
    v[0] = 10;
    v[1] = 20;
    v[2] = 30;
    compute_plm_coeff_2(v, k, a, minValue);
    for(int aa = 0; aa < 2; aa++)
    {
      printf("a[%d] = %d\n", aa, a[aa]);
    }
  }
}
void wrapper()
{
  float *b = new float[CUDASIZE];
  fillArray(b);
  float *dev_b;
  hipMalloc((void**)&dev_b, CUDASIZE*sizeof(float));
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(dev_b);
  /*
  double maxD = NPP_MAXABS_64F;
  double minD = NPP_MINABS_64F;

  double maxF = NPP_MAXABS_32F;
  double minF = NPP_MINABS_32F;

  double maxDV = std::numeric_limits<double>::max();
  double minDV = std::numeric_limits<double>::min();

  float maxFV = std::numeric_limits<float>::max();
  float minFV = std::numeric_limits<float>::min();

  printf("size of float = %ld\n", sizeof(float));
  printf("size of double = %ld\n", sizeof(double));

  if(maxD > maxDV)
    printf("maxD > maxDV");
  else if(maxD < maxDV)
    printf("maxD < maxDV");
  else
    printf("maxD = maxDV");
    printf("\n");
  if(minD > minDV)
    printf("minD > minDV");
  else if(minD < minDV)
    printf("minD < minDV");
  else
    printf("minD = minDV");
    printf("\n");
  */
}//end wrapper function

void fillArray(float *b)
{
  for(int a=0; a<CUDASIZE; a++)
  {
    b[a] = b[a] + CUDASIZE-a;
  }
}
