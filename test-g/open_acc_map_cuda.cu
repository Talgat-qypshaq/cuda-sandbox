#include "hip/hip_runtime.h"
#include "open_acc_map_h.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ));
static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
__global__ void cudaFunction(int *b)
{
  int index = threadIdx.x;
  //printf("CUDA [%d]: \n", index);
  if (index<CUDASIZE)
  {
    b[index] = b[index]+CUDASIZE;
  }
}
void cudaFunctionCallerWrapper(int a_1)
{
  printf("a_1: %d\n", a_1);
  int b[CUDASIZE];
  int *dev_b;
  HANDLE_ERROR( hipMalloc((void**)&dev_b, CUDASIZE * sizeof(int)) );
  for(int a=0; a<CUDASIZE; a++)
  {
    b[a] = a_1-a;
  }
  printf("before: b = %d\n", b[CUDASIZE-1]);
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cudaFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(int), hipMemcpyDeviceToHost);
  printf("after: b = %d\n", b[CUDASIZE-1]);
  hipFree(dev_b);
}

int main(void)
{
  printf("Hello World\n");
  int a = 1000;
  cudaFunctionCallerWrapper(a);
  return 0;
}
