#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void testFunction(int *dev_a)
{
  int thread = threadIdx.x+blockIdx.x*blockDim.x;
  if(thread < THREADS)
  {
    dev_a[thread] = dev_a[thread]*dev_a[thread];
    dev_a[thread] = testFunction(dev_a);
  }
}

__device__ int testFunction(int *dev_a)
{
    dev_a[thread] = dev_a[thread]*dev_a[thread];
    return dev_a;
}

int main()
{
  int a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE START 1\n");
  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\nBEFORE END 2\n");
  int *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(int));
  hipMemcpy(dev_a, a, THREADS*sizeof(int), hipMemcpyHostToDevice);
  testFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipMemcpy(a, dev_a, THREADS*sizeof(int), hipMemcpyDeviceToHost);

  printf("BEFORE START 1\n");
  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\nBEFORE END 2\n");
  hipFree(dev_a);
  return 0;
}
