#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void testFunction(float *dev_a)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    printf("dev[%d] = %.2f;\n", thread, dev_a[thread+2]);
    int c = 0;
    if (thread == 0)
    {
      int dev_b[2];
      dev_b[0] = 0;
      dev_b[1] = 1;
      c = dev_b[0] + dev_b[1];
    }
    if (thread == 1)
    {
      int dev_b[3];
      dev_b[0] = 0;
      dev_b[1] = 1;
      dev_b[2] = 2;
      c = dev_b[0] + dev_b[1] + dev_b[2];
    }
    if (thread == 2)
    {
      int dev_b[4];
      dev_b[0] = 0;
      dev_b[1] = 1;
      dev_b[2] = 2;
      dev_b[3] = 3;
      c = dev_b[0] + dev_b[1] + dev_b[2] + dev_b[3];
    }
    printf("c = %d;", c);
  }
}

int main()
{
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE START 1\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\nBEFORE END 1\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);
  testFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipFree(dev_a);
  printf("\nafter kernel.\n");
  return 0;
}
