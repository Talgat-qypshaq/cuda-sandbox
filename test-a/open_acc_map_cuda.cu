#include "hip/hip_runtime.h"
#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <utility>
#include <type_traits>
#include <stdio.h>
#include <stdlib.h>

#include "Vec4Simple.cu"

__global__ void cuda_global(int *dev_a, int *dev_b, int *dev_c)
{
  int i = threadIdx.x;
  if (i < CUDASIZE)
  {
    dev_c[i] = cuda_device_2(dev_a[i], dev_b[i]);
  }
}
void wrapper(int z)
{
  printf("STAGE 3\n");
  printf("z = %d\n", z);

  int a[CUDASIZE] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int b[CUDASIZE] = {11, 21, 31, 41, 51, 61, 71, 81, 91, 101};
  //int c[CUDASIZE];
  int c[CUDASIZE];

  int *dev_a;
  int *dev_b;
  //int *dev_c;

  int *dev_c;

  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMalloc((void**)&dev_b, CUDASIZE*sizeof(int));
  //hipMalloc((void**)&dev_c, CUDASIZE*sizeof(int));
  hipMalloc((void**)&dev_c, CUDASIZE*sizeof(int));

  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);

  cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, CUDASIZE*sizeof(int), hipMemcpyDeviceToHost);
  //printf("after: c = %d\n", c[CUDASIZE-3]);
  printf("after: c = %d\n", 2021);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
