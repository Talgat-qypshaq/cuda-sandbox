#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* cuda_device_1(int *dev_a, int *dev_b);
__device__ int cuda_device_2(int dev_a, int dev_b);

__device__ int* cuda_device_1(int *dev_a, int *dev_b)
{
  int i = threadIdx.x;
  if (i < CUDASIZE)
  {
    dev_a[i] = dev_a[i] + dev_b[i];
  }
  return dev_a;
}

__device__ int cuda_device_2(int dev_a, int dev_b)
{
  int i = threadIdx.x;
  int dev_c;
  if (i < CUDASIZE)
  {
    dev_c = dev_a + dev_b;
    printf("CUDA CODE\n");
  }
  return dev_c;
}
