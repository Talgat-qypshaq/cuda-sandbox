#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

extern __device__ Vec4Simple<int> cuda_device_1(int *dev_a, int *dev_b);

template <class T>
class Vec4Simple
{
  public:
   //T val[4] __attribute__((aligned(32)));
   T val[4];
   Vec4Simple() { }
   // Replicate scalar x across v.
   Vec4Simple(T x)
   {
      for(unsigned int i=0;i<4;i++)
         val[i]=x;
   }
   // Replicate 4 values across v.
   Vec4Simple(T a,T b,T c,T d)
   {
      val[0]=a;
      val[1]=b;
      val[2]=c;
      val[3]=d;
   }
   // Copy vector v.
   Vec4Simple(Vec4Simple const &x)
   {
      for(unsigned int i=0;i<4;i++)
         val[i]=x.val[i];
   }
   // Member function to load from array (unaligned)
   Vec4Simple & load(T const * p)
   {
      for(unsigned int i=0;i<4;i++)
         val[i]=p[i];
      return *this;
   }
   // Member function to load from array, aligned by 32
   Vec4Simple & load_a(T const * p)
   {
      return this->load(p);
   }

   Vec4Simple & insert(int i,T const &x)
   {
      val[i]=x;
      return *this;
   }
   // Member function to store into array (unaligned)
   void store(T * p) const
   {
      for(unsigned int i=0;i<4;i++)
         p[i]=val[i];
   }
   // Member function to store into array, aligned by 32
   void store_a(T * p) const
   {
      this->store(p);
   }
   /*
   Vec4Simple & operator = (Vec4Simple const & r)
   {
      for(unsigned int i=0;i<4;i++)
         val[i]=r.val[i];
      return *this;
   }
   */
   __device__ __host__ Vec4Simple & operator = (Vec4Simple const & r)
   {
      for(unsigned int i=0;i<4;i++)
         val[i]=r.val[i];
      return *this;
   }

   T operator [](int i) const
   {
     return val[i];
   }

   Vec4Simple operator++ (int)
   {
      Vec4Simple<T> temp (*this);
      for(unsigned int i=0;i<4;i++)
         val[i]++;
      return temp;
   }

   __device__ Vec4Simple<int> cuda_device_1(int *dev_a, int *dev_b)
   {
      Vec4Simple<T> temp (*this);
      int i = threadIdx.x;
      if (i < 4)
      {
        val[i] = dev_a[i] + dev_b[i];
      }
      return temp;
   }
};
