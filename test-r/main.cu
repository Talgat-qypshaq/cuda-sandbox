#include "hip/hip_runtime.h"
#include "header.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void globalFunction(int *dev_a)
{
  int thread = threadIdx.x + blockIdx.x*blockDim.x;
  if(thread < THREADS)
  {
    #ifdef ACC_SEMILAG_PLM
      int b[2];
      dev_a[thread] = dev_a[thread]*dev_a[thread];
    #endif
    #ifdef ACC_SEMILAG_PPM
      int b[3];
      dev_a[thread] = deviceFunction(dev_a[thread]);
    #endif
  }
}

int main()
{
  int a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("START\n");
  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\nEND\n");

  #ifdef ACC_SEMILAG_PLM
  int b[3];
  printf("ACC_SEMILAG_PLM");
  #endif
  #ifdef ACC_SEMILAG_PPM
  int b[3];
  printf("ACC_SEMILAG_PPM");
  #endif

  int *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(int));
  hipMemcpy(dev_a, a, THREADS*sizeof(int), hipMemcpyHostToDevice);
  globalFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipMemcpy(a, dev_a, THREADS*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\n");
  hipFree(dev_a);
  return 0;
}
