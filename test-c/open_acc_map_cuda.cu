#include "hip/hip_runtime.h"
#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__constant__ int dev_a;
__global__ void cudaFunction(int *b)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  //printf("CUDA [%d]: \n", index);
  if(index<CUDASIZE)
  {
    printf("dev_a = %d\n", dev_a);
    b[index] = b[index]+15;
  }
}

void wrapper(int c)
{
  printf("STAGE 3\n");
  printf("c = %d\n", c);

  int blockDataSizeTimesWID3 = 159;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_a), &blockDataSizeTimesWID3, sizeof(int));

  int b[CUDASIZE];
  int *dev_b;
  hipMalloc((void**)&dev_b, CUDASIZE*sizeof(int));
  for(int a=0; a<CUDASIZE; a++)
  {
    b[a] = c-a;
  }
  printf("before: b = %d\n", b[CUDASIZE-1]);
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cudaFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(int), hipMemcpyDeviceToHost);
  printf("after: b = %d\n", b[CUDASIZE-3]);
  hipFree(dev_b);
}
