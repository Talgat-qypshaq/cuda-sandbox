#include "hip/hip_runtime.h"
#include "open_acc_map_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__constant__ int dev_a;
__global__ void cudaFunction(int *b)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if(index<CUDASIZE)
  {
    b[index] = b[index]-3;
  }
}

void wrapper(int c)
{
  int b[CUDASIZE];
  for(int a=0;a<CUDASIZE;a++)
  {
    b[a] = c+a*c;
    printf("b[%d] = %d;\n", a, b[a]);
  }
  int *dev_b;
  hipMalloc((void**)&dev_b, CUDASIZE*sizeof(int));
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cudaFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(int), hipMemcpyDeviceToHost);
  printf("AFTER\n");
  for(int a=0;a<CUDASIZE;a++)
  {
    printf("b[%d] = %d;\n", a, b[a]);
  }
  hipFree(dev_b);
}
