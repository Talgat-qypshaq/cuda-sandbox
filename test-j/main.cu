#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void testFunction(float *dev_a, float *dev_b, float dev_c, float *dev_d)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    printf("dev[%d] = %.2f;\n", thread, dev_a[thread]);
    printf("b = %.2f;\n", *dev_b);
    printf("c 1 = %.2f;\n", dev_c);
    dev_c = dev_c*dev_c;
    printf("c 2 = %.2f;\n", dev_c);
    for(int i = 0; i<THREADS; i++)
    {
      printf("dev_d[%d] = %.2f; ", i, dev_d[i]);
    }
    printf("\nNOT WORKING!\n");
  }
}

int main()
{
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  float d[THREADS] = { 6, 7, 8, 9, 10 };
  printf("BEFORE START 1\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\nBEFORE END 2\n");
  printf("BEFORE START 2\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("d[%d] = %.2f; ", i, d[i]);
  }
  printf("\nBEFORE END 2\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);
  float b = 25;
  float *dev_b;
  hipMalloc((void**)&dev_b, sizeof(float));
  hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);
  float c = 77;
  testFunction<<<BLOCKS, THREADS>>>(dev_a, dev_b, c, d);
  hipFree(dev_a);
  hipFree(dev_b);
  printf("after kernel: c = %.2f;\n", c);
  return 0;
}
