#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void multiply(float *dev_a)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    dev_a[thread] = dev_a[thread]*dev_a[thread];
  }
}

int main()
{
  srand(static_cast<unsigned int>(time(0)));

  Solution s;
  int sparseMatrix[ROWS][COLS];

  for(int a=0;a<ROWS;a++)
  {
    for(int b=0;b<COLS;b++)
    {
      int c = s.getRandom();
      sparseMatrix[a][b] = c;
      printf("c = %d; ", c);
    }
    printf("\n");
  }

  int vector[COLS];
  for(int b=0;b<COLS;b++)
  {
    int d = s.getRandom();
    vector[b] = d;
    printf("d = %d; ", d);
  }
  printf("\n");

  float *dev_sparseMatrix;
  hipMalloc((void**)&dev_sparseMatrix, ROWS*COLS*sizeof(float));
  hipMemcpy(dev_sparseMatrix, sparseMatrix, ROWS*COLS*sizeof(float), hipMemcpyHostToDevice);

  float *dev_vector;
  hipMalloc((void**)&dev_vector, ROWS*COLS*sizeof(float));
  hipMemcpy(dev_vector, vector, ROWS*COLS*sizeof(float), hipMemcpyHostToDevice);

  multiply<<<BLOCKS, THREADS>>>(sparseMatrix, vector);
  
  hipFree(sparseMatrix);
  hipFree(vector);

  for(int c=0; c<3; c++)
    printf("columns[0].b[%d] = %d;\n", c, columns[0].b[c]);

  return 0;
}
