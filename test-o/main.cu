#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "Solution.h"

#define THREADS 3
#define BLOCKS 3

__global__ void multiply(int *dev_sparseMatrix, int *dev_vector, int *dev_result)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid == 1)
  {
    for(int a=0;a<BLOCKS;a++)
    {
      for(int b=0;b<THREADS;b++)
      {
        printf("SM[%d][%d] = %d; ", a, b, dev_sparseMatrix[tid]);
      }
      printf("\n");
    }
  }
  if(tid < BLOCKS * THREADS)
  {
    printf("tid = %d;\n", tid);
    if( dev_sparseMatrix[tid] == 0 || dev_vector[threadIdx.x] == 0 )
      dev_result[threadIdx.x] = dev_result[threadIdx.x] + 0;
    else
      dev_result[threadIdx.x] = dev_result[threadIdx.x] + dev_sparseMatrix[tid] * dev_vector[threadIdx.x];
  }
}

int main()
{
  srand(static_cast<unsigned int>(time(0)));

  Solution s;
  int sparseMatrix[BLOCKS][THREADS];
  printf("CPU START\n");
  for(int a=0;a<BLOCKS;a++)
  {
    for(int b=0;b<THREADS;b++)
    {
      sparseMatrix[a][b] = s.getRandom();
      printf("SM[%d][%d] = %d; ", a, b, sparseMatrix[a][b]);
    }
    printf("\n");
  }

  int vector[THREADS];
  int result[THREADS];
  for(int a=0;a<THREADS;a++)
  {
    result[a] = 0;
    vector[a] = s.getRandom();
    printf("vector[%d] = %d; ", a, vector[a]);
  }
  printf("\n");
  printf("CPU END\n");
  int *dev_sparseMatrix;
  hipMalloc((void**)&dev_sparseMatrix, BLOCKS*THREADS*sizeof(int));
  hipMemcpy(dev_sparseMatrix, sparseMatrix, BLOCKS*THREADS*sizeof(int), hipMemcpyHostToDevice);

  int *dev_vector;
  hipMalloc((void**)&dev_vector, BLOCKS*sizeof(int));
  hipMemcpy(dev_vector, vector, BLOCKS*sizeof(int), hipMemcpyHostToDevice);

  int *dev_result;
  hipMalloc((void**)&dev_result, THREADS*sizeof(int));
  printf("GPU START\n");
  multiply<<<BLOCKS, THREADS>>>(dev_sparseMatrix, dev_vector, dev_result);
  printf("GPU END\n");
  hipMemcpy(result, dev_result, THREADS*sizeof(int), hipMemcpyDeviceToHost);

  for(int c=0; c<3; c++)
    printf("result[%d] = %d;\n", c, result[c]);

  hipFree(sparseMatrix);
  hipFree(vector);
  hipFree(dev_result);

  return 0;
}
