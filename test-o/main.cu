#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "Solution.h"

#define THREADS 3
#define BLOCKS 3

__global__ void multiply(int *dev_sparseMatrix, int *dev_vector, int *dev_result)
{
  /*
  if(tid == 1)
  {
    for(int a=0;a<BLOCKS;a++)
    {
      for(int b=0;b<THREADS;b++)
      {
        printf("SM[%d][%d] = %d; ", a, b, dev_sparseMatrix[tid]);
      }
      printf("\n");
    }
  }

  while(tid < BLOCKS * THREADS)
  {
    temp += dev_sparseMatrix[tid] * dev_vector[threadIdx.x];
    tid += blockIdx.x * gridDim.x;
  }
  cacheResult[threadIdx.x] = temp;
  */
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //__shared__ int cacheResult[THREADS];
  //int temp = 0;
  if(tid < BLOCKS * THREADS)
  {
    //printf("tid = %d;\n", tid);
    if( dev_sparseMatrix[tid] != 0 && dev_vector[threadIdx.x] != 0 )
      dev_sparseMatrix[tid] = dev_sparseMatrix[tid] * dev_vector[threadIdx.x];
      //printf("dev_sparseMatrix[%d] = %d;\n", tid, dev_sparseMatrix[tid]);
  }
  __syncthreads();
  if(threadIdx.x < THREADS)
  {
    int i = 0;
    //printf("threadIdx.x = %d;\n", threadIdx.x);
    //printf("blockIdx.x = %d;\n", blockIdx.x);
    //printf("blockDim.x = %d;\n", blockDim.x);
    /*
    while(i < THREADS)
    {
      //dev_result[threadIdx.x] = dev_result[threadIdx.x] + dev_sparseMatrix[i+blockIdx.x*blockDim.x];
      //printf("i = %d; i+blockIdx.x*blockDim.x = %d\n", i, i+blockIdx.x*blockDim.x);
      //dev_result[threadIdx.x] = dev_result[threadIdx.x] + dev_sparseMatrix[i+blockIdx.x*blockDim.x];
      dev_sparseMatrix[i+blockIdx.x*blockDim.x] = dev_sparseMatrix[blockIdx.x*blockDim.x] + dev_sparseMatrix[i+blockIdx.x*blockDim.x];
      i++;
      __syncthreads();
      //printf("i = %d; dev_result[threadIdx.x] = %d;\n", i, dev_result[threadIdx.x]);
    }
    //__syncthreads();
    */
  }
  int cacheIndex = threadIdx.x;
  dev_result[cacheIndex] = temp;
  int i = blockDim.x/3;
  while(i!=0)
  {
    if(cacheIndex < i)
      dev_sparseMatrix[cacheIndex] += dev_sparseMatrix[cacheIndex+i];
    __syncthreads();
    i/=3;
  }
  if(tid<2)
  {
    if(threadIdx.x == 0)
    {
      dev_result[tid] = dev_sparseMatrix[threadIdx.x];
    }
  }

  //if(threadIdx.x < THREADS)
  //{
  //  dev_result[threadIdx.x] = cacheResult[threadIdx.x];
  //}
  /*
  if(tid < BLOCKS * THREADS)
  {
    printf("tid = %d;\n", tid);
    if( dev_sparseMatrix[tid] == 0 || dev_vector[threadIdx.x] == 0 )
      dev_result[threadIdx.x] = dev_result[threadIdx.x] + 0;
    else
      dev_result[threadIdx.x] = dev_result[threadIdx.x] + dev_sparseMatrix[tid] * dev_vector[threadIdx.x];
    __syncthreads();
  }
  */
}

int main()
{
  srand(static_cast<unsigned int>(time(0)));

  Solution s;
  int sparseMatrix[BLOCKS][THREADS];
  printf("CPU START\n");
  for(int a=0;a<BLOCKS;a++)
  {
    for(int b=0;b<THREADS;b++)
    {
      sparseMatrix[a][b] = s.getRandom();
      printf("SM[%d][%d] = %d; ", a, b, sparseMatrix[a][b]);
    }
    printf("\n");
  }

  int vector[THREADS];
  int result[THREADS];
  for(int a=0;a<THREADS;a++)
  {
    result[a] = 0;
    vector[a] = s.getRandom();
    printf("vector[%d] = %d; ", a, vector[a]);
  }
  printf("\n");
  printf("CPU END\n");
  int *dev_sparseMatrix;
  hipMalloc((void**)&dev_sparseMatrix, BLOCKS*THREADS*sizeof(int));
  hipMemcpy(dev_sparseMatrix, sparseMatrix, BLOCKS*THREADS*sizeof(int), hipMemcpyHostToDevice);

  int *dev_vector;
  hipMalloc((void**)&dev_vector, BLOCKS*sizeof(int));
  hipMemcpy(dev_vector, vector, BLOCKS*sizeof(int), hipMemcpyHostToDevice);

  int *dev_result;
  hipMalloc((void**)&dev_result, THREADS*sizeof(int));
  multiply<<<BLOCKS, THREADS>>>(dev_sparseMatrix, dev_vector, dev_result);
  hipMemcpy(result, dev_result, THREADS*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sparseMatrix, dev_sparseMatrix, THREADS*sizeof(int), hipMemcpyDeviceToHost);
  for(int c=0; c<3; c++)
    printf("result[%d] = %d;\n", c, result[c]);

  for(int a=0; a<3; a++)
  {
    for(int b=0; b<3; b++)
    {
        printf("C SM[%d][%d] = %d; ", a, b, sparseMatrix[a][b]);
    }
    printf("\n");
  }
  hipFree(sparseMatrix);
  hipFree(vector);
  hipFree(dev_result);

  return 0;
}
