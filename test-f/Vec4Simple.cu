#include "open_acc_map_header.cuh"

#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

template <typename T>
CUDA_HOSTDEV Vec4Simple<T>::Vec4Simple() { }
template <typename T>
CUDA_HOSTDEV Vec4Simple<T>::Vec4Simple(T *p) : _p(p) {}
template <typename T>
CUDA_HOSTDEV T Vec4Simple<T>::getSquare(T b)
{ return b*b; }
template <typename T>
CUDA_HOSTDEV Vec4Simple<T>::Vec4Simple(T x)
{
  for(unsigned int i=0;i<4;i++)
    val[i]=x;
}
template <typename T>
CUDA_HOSTDEV Vec4Simple<T>::Vec4Simple(T a,T b,T c,T d)
{
   val[0]=a;
   val[1]=b;
   val[2]=c;
   val[3]=d;
}
template <typename T>
CUDA_HOSTDEV Vec4Simple<T>::Vec4Simple(Vec4Simple const &x)
{
   for(unsigned int i=0;i<4;i++)
      val[i]=x.val[i];
}

// Member function to load from array (unaligned)
template <typename T>
CUDA_HOSTDEV Vec4Simple<T> & Vec4Simple<T>::load(T const * p)
{
   for(unsigned int i=0;i<4;i++)
      val[i]=p[i];
   return *this;
}

// Member function to load from array, aligned by 32
template <typename T>
CUDA_HOSTDEV Vec4Simple<T> & Vec4Simple<T>::load_a(T const * p)
{
   return this->load(p);
}

template <typename T>
CUDA_HOSTDEV Vec4Simple<T> & Vec4Simple<T>::insert(int i,T const &x)
{
   val[i]=x;
   return *this;
}

// Member function to store into array (unaligned)
template <typename T>
CUDA_HOSTDEV void Vec4Simple<T>::store(T * p) const
{
   for(unsigned int i=0;i<4;i++)
      p[i]=val[i];
}

// Member function to store into array, aligned by 32
template <typename T>
CUDA_HOSTDEV void Vec4Simple<T>::store_a(T * p) const
{
   this->store(p);
}

template <typename T>
CUDA_HOSTDEV Vec4Simple<T> & Vec4Simple<T>::operator = (Vec4Simple<T> const & r)
{
   for(unsigned int i=0;i<4;i++)
      val[i]=r.val[i];
   return *this;
}

template <typename T>
CUDA_HOSTDEV T Vec4Simple<T>::operator [](int i) const
{
   return val[i];
}

template <typename T>
CUDA_HOSTDEV Vec4Simple<T> Vec4Simple<T>::operator++ (int)
{
   Vec4Simple<T> temp (*this);
   for(unsigned int i=0;i<4;i++)
      val[i]++;
   return temp;
}

template class Vec4Simple<int>;

template <class T>
static inline Vec4Simple<T> truncate_to_int(Vec4Simple<T> const & a)
{
  return Vec4Simple<int>(a.val[0], a.val[1], a.val[2], a.val[3]);
}
