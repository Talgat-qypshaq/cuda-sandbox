#include "hip/hip_runtime.h"
#include "cuda_header.cuh"
#include "open_acc_map_header.cuh"
#include "vec.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <utility>
#include <type_traits>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_global(int *dev_a)
{
  int i = threadIdx.x;
  if (i == 0)
  {
    printf("CUDA START\n");
    Veci *c = new Veci[CUDASIZE];
    const Veci i_indices = Veci(0, 1, 2, 3);
    for (int d = 0; d < CUDASIZE; d++)
    {
      printf("i_indices[%d] = %d; ", d, i_indices[d]);
    }
    printf("\n");
    /*
    for (int a = 0; a < CUDASIZE; a++)
    {
      for (int b = 0; b < CUDASIZE; b++)
      {
        printf("c[%d][%d] = %d; ", a, b, c[a][b]);
      }
      printf("\n");
    }
    */
    printf("CUDA END\n");
  }
}
void wrapper()
{
  int *a = new int[CUDASIZE];
  int *dev_a;
  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  const Veci i_indices = Veci(0, 1, 2, 3);
  printf("C++ START\n");
  for (int d = 0; d < CUDASIZE; d++)
  {
    printf("i_indices[%d] = %d; ", d, i_indices[d]);
  }
  printf("\n");
  /*
  Veci *c = new Veci[CUDASIZE];
  for (int a = 0; a < CUDASIZE; a++)
  {
    c[a] = a*3;
  }
  for (int a = 0; a < CUDASIZE; a++)
  {
    for (int b = 0; b < CUDASIZE; b++)
    {
      printf("c[%d][%d] = %d; ", a, b, c[a][b]);
    }
    printf("\n");
  }
  */
  printf("C++ END\n");
  cuda_global<<<BLOCKS, THREADS>>>(dev_a);

  hipMemcpy(a, dev_a, CUDASIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_a);
  printf("CUDA END\n");
}
