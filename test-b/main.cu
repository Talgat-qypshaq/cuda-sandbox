#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits>
using namespace std;
#define NPP_MAXABS_32F ( 3.402823466e+38f )
#define NPP_MINABS_32F ( 1.175494351e-38f )

#define NPP_MAXABS_64F ( 1.7976931348623158e+308 )
#define NPP_MINABS_64F ( 2.2250738585072014e-308 )

#define THREADS 5
#define BLOCKS 10

typedef double apple;
typedef float orange;

__global__ void testFunction(float *dev_a, float *dev_b, orange *dev_c)
{
  int thread = threadIdx.x;
  if(thread < THREADS)
  {
    if(thread < NPP_MAXABS_32F)
      dev_a[thread] = NPP_MAXABS_32F;
    else
      dev_a[thread] = NPP_MINABS_32F;
    if (thread == 4)
    {
      dev_a[thread] = *dev_b;
      *dev_b = dev_a[thread]*2;
      dev_a[thread-1] = *dev_c;
      *dev_c = dev_a[thread-1]*3;
    }
  }
}

int main()
{
  printf("\nMAIN START\n");
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE LOOP\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("AFTER LOOP\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);

  float b = 25;
  float *dev_b;
  hipMalloc((void**)&dev_b, sizeof(float));
  hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);

  orange c = 77;
  orange *dev_c;
  hipMalloc((void**)&dev_c, sizeof(orange));
  hipMemcpy(dev_c, &c, sizeof(orange), hipMemcpyHostToDevice);

  testFunction<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_c);

  hipMemcpy(a, dev_a, THREADS*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&b, dev_b, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&c, dev_c, sizeof(orange), hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  printf("\nAFTER CUDA FREE\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\nEND\n");
  printf("b = %.2f; \n", b);
  printf("c = %.2f; \n", c);

  orange d = 12.3;
  apple e = 23.4;
  printf("d = %.2f\ne = %.2f\n", d, e);
  printf("MAX: %.2f\n", NPP_MAXABS_32F);
  printf("MAX FLOAT: %.2f\n", numeric_limits<float>::max());
  printf("MAX DOUBLE: %.2f\n", numeric_limits<double>::max());

  return 0;
}
